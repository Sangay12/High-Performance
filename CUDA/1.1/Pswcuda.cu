#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
__device__ int is_a_match(char *attempt) {
char password_a[] = "KJ1231";
char password_b[] = "IP9812";
char password_c[] = "MM3212";
char password_d[] = "VC0054";
char *a = attempt;
char *b = attempt;
char *c = attempt;
char *d = attempt;
char *p1 = password_a;
char *p2 = password_b;
char *p3 = password_c;
char *p4 = password_d;
while(*a == *p1) {if(*a == '\0')
{
printf("Found password: %s\n",password_a);
break;
}
a++;
p1++;
}
while(*b == *p2) {
if(*b == '\0')
{
printf("Found password: %s\n",password_b);
break;
}
b++;
p2++;
}
while(*c == *p3) {
if(*c == '\0')
{
printf("Found password: %s\n",password_c);
break;
}
c++;
p3++;
}
while(*d == *p4) {
if(*d == '\0')
{
printf("Found password: %s\n",password_d);
return 1;
}
d++;
p4++;
}
return 0;
}

/****************************************************************************
The kernel function assume that there will be only one thread and uses nested loops
to generate all possible passwords and test whether they match the hidden password.
*****************************************************************************/

__global__ void kernel() {
char k,l,m,n;
char password[7];
password[6] = '\0';
int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
password[0] = firstValue;
password[1] = secondValue;
for(k='0'; k<='9'; k++){
for(l='0'; l<='9'; l++){
for(m='0'; m<='9'; m++){
for(n='0'; n<='9'; n++){
password[2] = k;
password[3] = l;
password[4] = m;
password[5] = n;
if(is_a_match(password)) {
//printf("Success");
}
else {
//printf("tried: %s\n", password);
}
}
}
}
}
}
int time_difference(struct timespec *start,
struct timespec *finish,
long long int *difference) {
long long int ds = finish->tv_sec - start->tv_sec;
long long int dn = finish->tv_nsec - start->tv_nsec;
if(dn < 0 ) {ds--;
dn += 1000000000;
}
*difference = ds * 1000000000 + dn;
return !(*difference > 0);
}
int main() {
struct timespec start, finish;
long long int time_elapsed;
clock_gettime(CLOCK_MONOTONIC, &start);
kernel <<<26,26>>>();
hipDeviceSynchronize();
clock_gettime(CLOCK_MONOTONIC, &finish);
time_difference(&start, &finish, &time_elapsed);
printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
(time_elapsed/1.0e9));
return 0;
}
