#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
 Compile and run with:
    nvcc -o Two_inital_4_digits Two_inital_4_digits.cu
   
    To run
    ./Two_inital_4_digits

  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "DP2323";
  char plain_password2[] = "BV1212";
  char plain_password3[] = "WE1723";
  char plain_password4[] = "PO0912";

  char *e = attempt;
  char *f = attempt;
  char *g = attempt;
  char *h = attempt;
  char *p1 = plain_password1;
  char *p2 = plain_password2;
  char *p3 = plain_password3;
  char *p4 = plain_password4;

  while(*e == *p1) { 
   if(*e == '\0') 
    {
	printf("Found password: %s\n",plain_password1);
      break;
    }

    e++;
    p1++;
  }
	
  while(*f == *p2) { 
   if(*f == '\0') 
    {
	printf("Found password: %s\n",plain_password2);
      break;
    }

    f++;
    p2++;
  }

  while(*g == *p3) { 
   if(*g == '\0') 
    {
	printf("Found password: %s\n",plain_password3);
      break;
    }

    g++;
    p3++;
  }

  while(*h == *p4) { 
   if(*h == '\0') 
    {
	printf("Found password: %s\n",plain_password4);
      return 1;
    }

    h++;
    p4++;
  }
  return 0;

}


__global__ void  kernel() {
char u,v,w,x;
  
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i; 
char secondValue = j; 
    
password[0] = firstValue;
password[1] = secondValue;
	for(u='0'; u<='9'; u++){
	  for(v='0'; v<='9'; v++){
            for(w='0'; w<='9'; w++){
               for(x='0'; x<='9'; x++){
	   
	        password[2] = u;
	        password[3] = v;
                password[4] = w;
                password[5] = x;
          is_a_match(password);
	      }
	   }
	}
      }
   }
   

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}


