#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
 Compile and run with:
    nvcc -o Pswcuda Pswcuda.cu
   
    To run
    ./Pswcuda

  Dr Kevan Buckley, University of Wolverhampton, 2019
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
char passw_a[] = "KJ1231";
char passw_b[] = "IP9812";
char passw_c[] = "MM3212";
char passw_d[] = "VC0054";
char *q = attempt;
char *r = attempt;
char *t = attempt;
char *o = attempt;
char *pa1 = passw_a;
char *pa2 = passw_b;
char *pa3 = passw_c;
char *pa4 = passw_d;
while(*q == *pa1) {if(*q == '\0')
{
printf("Found password: %s\n",passw_a);
break;
}
q++;
pa1++;
}
while(*r == *pa2) {
if(*r == '\0')
{
printf("Found password: %s\n",passw_b);
break;
}
r++;
pa2++;
}
while(*t == *pa3) {
if(*t == '\0')
{
printf("Found password: %s\n",passw_c);
break;
}
t++;
pa3++;
}
while(*o == *pa4) {
if(*o == '\0')
{
printf("Found password: %s\n",passw_d);
return 1;
}
o++;
pa4++;
}
return 0;
}

/****************************************************************************
The kernel function assume that there will be only one thread and uses nested loops
to generate all possible passwords and test whether they match the hidden password.
*****************************************************************************/

__global__ void kernel() {
char v,b,m,n;
char password[7];
password[6] = '\0';
int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
password[0] = firstValue;
password[1] = secondValue;
for(v='0'; v<='9'; v++){
for(b='0'; b<='9'; b++){
for(m='0'; m<='9'; m++){
for(n='0'; n<='9'; n++){
password[2] = v;
password[3] = b;
password[4] = m;
password[5] = n;
if(is_a_match(password)) {
//printf("Success");
}
else {
//printf("tried: %s\n", password);
}
}
}
}
}
}
int time_difference(struct timespec *start,
struct timespec *finish,
long long int *difference) {
long long int ds = finish->tv_sec - start->tv_sec;
long long int dn = finish->tv_nsec - start->tv_nsec;
if(dn < 0 ) {ds--;
dn += 1000000000;
}
*difference = ds * 1000000000 + dn;
return !(*difference > 0);
}
int main() {
struct timespec start, finish;
long long int time_elapsed;
clock_gettime(CLOCK_MONOTONIC, &start);
kernel <<<26,26>>>();
hipDeviceSynchronize();
clock_gettime(CLOCK_MONOTONIC, &finish);
time_difference(&start, &finish, &time_elapsed);
printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
(time_elapsed/1.0e9));
return 0;
}
